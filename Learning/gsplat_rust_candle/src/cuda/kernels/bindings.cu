#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math.h>
#include <tuple>

namespace cg = cooperative_groups;

// Une des fonctions CUDA utilisées par programme était dans bindins.cu donc copie de ce dernier en enlevent les transformation en torch::Tensor
__global__ void compute_cov2d_bounds_kernel(
    const unsigned num_pts, const float* __restrict__ covs2d, float* __restrict__ conics, float* __restrict__ radii
) {
    unsigned row = cg::this_grid().thread_rank();
    if (row >= num_pts) {
        return;
    }
    int index = row * 3;
    float3 conic;
    float radius;
    float3 cov2d{
        (float)covs2d[index], (float)covs2d[index + 1], (float)covs2d[index + 2]
    };
    compute_cov2d_bounds(cov2d, conic, radius);
    conics[index] = conic.x;
    conics[index + 1] = conic.y;
    conics[index + 2] = conic.z;
    radii[row] = radius;
}